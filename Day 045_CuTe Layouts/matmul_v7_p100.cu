#include "hip/hip_runtime.h"
/*
 * GEMM Kernel with cuBLAS Benchmarking
 * Tests multiple matrix sizes: 512, 1024, 2048, 4096, 8192
 * 
 * GEMM Kernel Assumptions and Requirements:
 * 
 * 1. Matrix Dimension Divisibility:
 *    - M % BM_dim == 0 (No partial tiles in M dimension)
 *    - N % BN_dim == 0 (No partial tiles in N dimension)
 *    - K % BK_dim == 0 (No partial tiles in K dimension)
 * 
 * 2. Thread Configuration:
 *    - NUM_THREADS == (BN_dim/TN_dim) * (BM_dim/TM_dim) (Exact thread count for tile coverage)
 *    - blockDim.x == NUM_THREADS (1D thread block organization)
 *    - NUM_THREADS <= 1024 (Maximum threads per block on modern GPUs)
 * 
 * 3. Memory Alignment and Coalescing:
 *    - BK_dim % 32 == 0 (For bank conflict-free shared memory access)
 *    - BN_dim % 32 == 0 (For perfect global memory coalescing)
 *    - BM_dim % 4 == 0 (For vectorized loads of A matrix)
 *    - All global memory pointers (A,B,C) are 16-byte aligned
 * 
 * 4. Shared Memory Constraints:
 *    - (BM_dim*BK_dim + BK_dim*BN_dim)*sizeof(float) <= 48KB (P100 shared memory limit)
 *    - BM_dim*BK_dim % 4 == 0 (For vectorized stores to shared memory)
 * 
 * 5. Register Usage:
 *    - TM_dim % 4 == 0 (For vectorized register loads)
 *    - TN_dim % 4 == 0 (For vectorized register stores)
 * 
 * 6. Kernel Parameters:
 *    - BM_dim >= TM_dim && BM_dim % TM_dim == 0
 *    - BN_dim >= TN_dim && BN_dim % TN_dim == 0
 *    - BK_dim >= 4 (Minimum for vectorized loads)
 * 
 * Note: These constraints are automatically verified via static_assert in the kernel.
 *       For partial tile support, additional boundary checks would be required.
 */

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <vector>
#include <cassert>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t err = (call); \
        if (err != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << err << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)



// Vectorized Memory Load
template<unsigned int TILE_ROWS,
         unsigned int TILE_COLS,
         unsigned int NUM_THREADS>
__device__ __forceinline__ void tileMemcpyVectorized(
    float* __restrict__ src,
    float* __restrict__ dst,
    const unsigned int src_stride
)
{
    // Reinterpret as float4 pointers for vectorized access
    float4* src_float4 = reinterpret_cast<float4*>(src);
    float4* dst_float4 = reinterpret_cast<float4*>(dst);
    
    // Calculate vectorized strides (in terms of float4 elements)
    const unsigned int src_stride_vectorized = src_stride / 4;
    

    // Verify vectorization requirements
    static_assert(TILE_COLS % 4 == 0, "TILE_COLS must be divisible by 4 for vectorization");
    static_assert(sizeof(float4) == 16, "float4 must be 16 bytes");
    
    // Calculate thread layout parameters
    constexpr unsigned int TILE_COLS_VECTORIZED = TILE_COLS / 4;
    static_assert(NUM_THREADS % TILE_COLS_VECTORIZED == 0, 
                 "Thread count must be divisible by vectorized columns");
    
    constexpr unsigned int ROW_STEP = NUM_THREADS / TILE_COLS_VECTORIZED;
    constexpr unsigned int NUM_ITERS = TILE_ROWS / ROW_STEP;

    // Calculate thread position in tile grid
    const unsigned int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int thread_row = thread_idx / TILE_COLS_VECTORIZED;
    const unsigned int thread_col = thread_idx % TILE_COLS_VECTORIZED;

    // Vectorized copy loop
    #pragma unroll
    for (unsigned int i = 0; i < NUM_ITERS; i++)
    {
        // Read vectorized data from source
        float4 vec = src_float4[thread_row * src_stride_vectorized + thread_col];
        
        // Write vectorized data to destination
        dst_float4[thread_row * TILE_COLS_VECTORIZED + thread_col] = vec;
        
        // Move to next row segment
        thread_row += ROW_STEP;
    }
}

// Loading from Shared Memory to Registers 
template <int TM_dim, int TN_dim, int BK_dim, int BN_dim>
__device__ __forceinline__ void load_matrix_tiles_to_registers(
    const float* __restrict__ A_block_smem,
    const float* __restrict__ B_block_smem,
    float* __restrict__ A_register,  // Use pointer (saves 1 register)
    float* __restrict__ B_register,
    const int thread_row,
    const int thread_col,
    const int k_inner
) {
    // Load A fragment (column-major)
    #pragma unroll
    for (int tm = 0; tm < TM_dim; tm++) {
        A_register[tm] = A_block_smem[(thread_row + tm) * BK_dim + k_inner];
    }
    
    // Load B fragment (row-major)
    #pragma unroll
    for (int tn = 0; tn < TN_dim; tn++) {
        B_register[tn] = B_block_smem[k_inner * BN_dim + thread_col + tn];
    }
}

template <
    unsigned int BM_dim,
    unsigned int BN_dim,
    unsigned int BK_dim,
    unsigned int TM_dim,
    unsigned int TN_dim,
    unsigned int NUM_THREADS
>
__global__ void sgemm_v2_kernel(
    float* __restrict__ A,
    float* __restrict__ B,
    float* __restrict__ C,
    const float alpha,
    const float beta,
    const unsigned int M,
    const unsigned int N,
    const unsigned int K
) {
    // Validate thread configuration
    static_assert((BN_dim / TN_dim) * (BM_dim / TM_dim) == NUM_THREADS, 
                 "Invalid thread count");
    // For Coalescing
    //static_assert(BK_dim % 32 == 0, "BK must be multiple of 32 for coalescing");
    //static_assert(BN_dim % 32 == 0, "BN must be multiple of 32 for coalescing");
                 
    // 1. Leading dimensions
    const unsigned int A_stride = K;
    const unsigned int B_stride = N;
    const unsigned int C_stride = N;

    // 2. Block index
    const unsigned int block_m = blockIdx.y;
    const unsigned int block_n = blockIdx.x;

    // 3. Thread index
    const unsigned int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    const unsigned int threads_per_block_row = BN_dim / TN_dim;
    const unsigned int thread_row = (thread_id / threads_per_block_row) * TM_dim;
    const unsigned int thread_col = (thread_id % threads_per_block_row) * TN_dim;

    // 4. Shared memory allocation with double buffers
    extern __shared__ float shmem[];
    float* A_block_smem[2] = { shmem, shmem + BM_dim * BK_dim };
    float* B_block_smem[2] = { 
        shmem + 2 * BM_dim * BK_dim, 
        shmem + 2 * BM_dim * BK_dim + BK_dim * BN_dim 
    };

    // 5. Register allocation with double buffers
    float A_register[2][TM_dim] = {{0.}};
    float B_register[2][TN_dim] = {{0.}};
    float acc_register[TM_dim][TN_dim] = {{0.}};

    // 6. Prefetch first block
    unsigned int block_k = 0;
    unsigned int current_block_buffer = 0;
    unsigned int next_block_buffer = 1;
    
    float* A_block_gmem0 = &A[block_m * BM_dim * A_stride + 0 * BK_dim];
    float* B_block_gmem0 = &B[0 * BK_dim * B_stride + block_n * BN_dim];
    
    tileMemcpyVectorized<BM_dim, BK_dim, NUM_THREADS>(
        A_block_gmem0, A_block_smem[current_block_buffer], A_stride);
    tileMemcpyVectorized<BK_dim, BN_dim, NUM_THREADS>(
        B_block_gmem0, B_block_smem[current_block_buffer], B_stride);
    __syncthreads();
    
    // 7. Load first fragment for first block
    load_matrix_tiles_to_registers<TM_dim, TN_dim, BK_dim, BN_dim>(
        A_block_smem[current_block_buffer], 
        B_block_smem[current_block_buffer],
        A_register[0], 
        B_register[0],
        thread_row, thread_col, 0);

    // 8. Main computation loop
    const unsigned int num_block_tiles_k = K / BK_dim;
    for (block_k = 0; block_k < num_block_tiles_k; ++block_k) {
        current_block_buffer = block_k % 2;
        next_block_buffer = (block_k + 1) % 2;

        // Prefetch next block
        __syncthreads();
        if (block_k < num_block_tiles_k - 1) {
            float* A_block_gmem_next = &A[block_m * BM_dim * A_stride + (block_k+1) * BK_dim];
            float* B_block_gmem_next = &B[(block_k+1) * BK_dim * B_stride + block_n * BN_dim];
            tileMemcpyVectorized<BM_dim, BK_dim, NUM_THREADS>(
                A_block_gmem_next, A_block_smem[next_block_buffer], A_stride);
            tileMemcpyVectorized<BK_dim, BN_dim, NUM_THREADS>(
                B_block_gmem_next, B_block_smem[next_block_buffer], B_stride);
            __syncthreads();
        }

        // Inner loop over BK_dim
        for (unsigned int k_inner = 0; k_inner < BK_dim; ++k_inner) {
            unsigned int current_frag_buffer = k_inner % 2;
            unsigned int next_frag_buffer = (k_inner + 1) % 2;

            // Prefetch next fragment
            if (k_inner < BK_dim - 1) {
                load_matrix_tiles_to_registers<TM_dim, TN_dim, BK_dim, BN_dim>(
                    A_block_smem[current_block_buffer], 
                    B_block_smem[current_block_buffer],
                    A_register[next_frag_buffer], 
                    B_register[next_frag_buffer],
                    thread_row, thread_col, k_inner + 1);
            }

            // Compute with current fragment
            #pragma unroll
            for (unsigned int tm = 0; tm < TM_dim; ++tm) {
                #pragma unroll
                for (unsigned int tn = 0; tn < TN_dim; ++tn) {
                    acc_register[tm][tn] += 
                        A_register[current_frag_buffer][tm] * 
                        B_register[current_frag_buffer][tn];
                }
            }
        }

        // Prefetch first fragment for next block
        if (block_k < num_block_tiles_k - 1) {
            load_matrix_tiles_to_registers<TM_dim, TN_dim, BK_dim, BN_dim>(
                A_block_smem[next_block_buffer], 
                B_block_smem[next_block_buffer],
                A_register[0], 
                B_register[0],
                thread_row, thread_col, 0);
        }
    }

    // 9. Final output with alpha/beta scaling
    float* C_block_gmem = &C[block_m * BM_dim * C_stride + block_n * BN_dim];
    #pragma unroll
    for (unsigned int tm = 0; tm < TM_dim; ++tm) {
        #pragma unroll
        for (unsigned int tn = 0; tn < TN_dim; ++tn) {
            unsigned int row = thread_row + tm;
            unsigned int col = thread_col + tn;
            if (row < M && col < N) {
                float c_val = C_block_gmem[row * C_stride + col];
                C_block_gmem[row * C_stride + col] = 
                    alpha * acc_register[tm][tn] + beta * c_val;
            }
        }
    }
}

// CPU reference GEMM implementation
void cpu_gemm(
    float* A, float* B, float* C,
    float alpha, float beta,
    unsigned int M, unsigned int N, unsigned int K
) {
    for (unsigned int m = 0; m < M; ++m) {
        for (unsigned int n = 0; n < N; ++n) {
            float sum = 0.0f;
            for (unsigned int k = 0; k < K; ++k) {
                sum += A[m * K + k] * B[k * N + n];
            }
            C[m * N + n] = alpha * sum + beta * C[m * N + n];
        }
    }
}

// Initialize matrix with random values
void init_matrix(float* mat, unsigned int size) {
    for (unsigned int i = 0; i < size; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Compare two matrices
bool verify_results(float* gpu, float* cpu, unsigned int size, float epsilon = 1e-3) {
    unsigned int mismatches = 0;
    float max_error = 0.0f;
    
    for (unsigned int i = 0; i < size; ++i) {
        float error = fabs(gpu[i] - cpu[i]);
        max_error = fmax(max_error, error);
        
        if (error > epsilon) {
            mismatches++;
            if (mismatches <= 5) { // Print first 5 mismatches
                std::cerr << "Mismatch at index " << i 
                          << ": GPU=" << gpu[i] << ", REF=" << cpu[i] 
                          << ", Error=" << error << std::endl;
            }
        }
    }
    
    std::cout << "Max error: " << max_error << ", Mismatches: " << mismatches 
              << "/" << size << std::endl;
    
    return mismatches == 0;
}

// Calculate GFLOPS
double calculate_gflops(unsigned int M, unsigned int N, unsigned int K, float time_ms) {
    double flops = 2.0 * M * N * K; // 2 operations per element (multiply + add)
    return (flops / 1e9) / (time_ms / 1000.0);
}

// Benchmark function
void benchmark_gemm(unsigned int M, unsigned int N, unsigned int K) {
    std::cout << "\n" << std::string(80, '=') << std::endl;
    std::cout << "Benchmarking GEMM: M=" << M << ", N=" << N << ", K=" << K << std::endl;
    std::cout << std::string(80, '=') << std::endl;

    // Tile dimensions - adjust based on matrix size
    constexpr unsigned int BM_dim = 32*2;
    constexpr unsigned int BN_dim = 32*2;
    constexpr unsigned int BK_dim = 8;
    constexpr unsigned int TM_dim = 8;
    constexpr unsigned int TN_dim = 8;
    constexpr unsigned int NUM_THREADS = BM_dim * BN_dim / (TM_dim * TN_dim);

    // Check if dimensions are compatible
    if (M % BM_dim != 0 || N % BN_dim != 0 || K % BK_dim != 0) {
        std::cout << "Skipping: Matrix dimensions not compatible with tile sizes" << std::endl;
        return;
    }

    // Allocate host memory
    std::vector<float> h_A(M * K);
    std::vector<float> h_B(K * N);
    std::vector<float> h_C_custom(M * N);
    std::vector<float> h_C_cublas(M * N);
    std::vector<float> h_C_original(M * N);

    // Initialize matrices
    srand(42); // Fixed seed for reproducible results
    init_matrix(h_A.data(), M * K);
    init_matrix(h_B.data(), K * N);
    init_matrix(h_C_original.data(), M * N);
    
    // Copy initial C matrix
    std::copy(h_C_original.begin(), h_C_original.end(), h_C_custom.begin());
    std::copy(h_C_original.begin(), h_C_original.end(), h_C_cublas.begin());

    // Allocate device memory
    float *d_A, *d_B, *d_C_custom, *d_C_cublas;
    CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C_custom, M * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C_cublas, M * N * sizeof(float)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_custom, h_C_custom.data(), M * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_cublas, h_C_cublas.data(), M * N * sizeof(float), hipMemcpyHostToDevice));

    const float alpha = 1.0f;
    const float beta = 0.5f;
    const int num_runs = 10;

    // Create events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Warm up GPU
    for (int i = 0; i < 3; ++i) {
        dim3 grid(N / BN_dim, M / BM_dim);
        dim3 block(BN_dim / TN_dim, BM_dim / TM_dim);
        size_t shmem_size = 2 * (BM_dim * BK_dim + BK_dim * BN_dim) * sizeof(float);
        
        sgemm_v2_kernel<BM_dim, BN_dim, BK_dim, TM_dim, TN_dim, NUM_THREADS>
            <<<grid, block, shmem_size>>>(d_A, d_B, d_C_custom, alpha, beta, M, N, K);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    // Benchmark custom kernel
    float custom_total_time = 0.0f;
    for (int run = 0; run < num_runs; ++run) {
        // Reset C matrix
        CUDA_CHECK(hipMemcpy(d_C_custom, h_C_original.data(), M * N * sizeof(float), hipMemcpyHostToDevice));
        
        dim3 grid(N / BN_dim, M / BM_dim);
        dim3 block(BN_dim / TN_dim, BM_dim / TM_dim);
        size_t shmem_size = 2 * (BM_dim * BK_dim + BK_dim * BN_dim) * sizeof(float);
        
        CUDA_CHECK(hipEventRecord(start));
        sgemm_v2_kernel<BM_dim, BN_dim, BK_dim, TM_dim, TN_dim, NUM_THREADS>
            <<<grid, block, shmem_size>>>(d_A, d_B, d_C_custom, alpha, beta, M, N, K);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        
        float milliseconds;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        custom_total_time += milliseconds;
    }
    
    float custom_avg_time = custom_total_time / num_runs;
    double custom_gflops = calculate_gflops(M, N, K, custom_avg_time);

    // Create cuBLAS handle
    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));

    // Warm up cuBLAS
    for (int i = 0; i < 3; ++i) {
        CUBLAS_CHECK(hipblasSgemm(
            cublas_handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, M, K,
            &alpha,
            d_B, N,
            d_A, K,
            &beta,
            d_C_cublas, N
        ));
    }
    CUDA_CHECK(hipDeviceSynchronize());

    // Benchmark cuBLAS
    float cublas_total_time = 0.0f;
    for (int run = 0; run < num_runs; ++run) {
        // Reset C matrix
        CUDA_CHECK(hipMemcpy(d_C_cublas, h_C_original.data(), M * N * sizeof(float), hipMemcpyHostToDevice));
        
        CUDA_CHECK(hipEventRecord(start));
        CUBLAS_CHECK(hipblasSgemm(
            cublas_handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, M, K,
            &alpha,
            d_B, N,
            d_A, K,
            &beta,
            d_C_cublas, N
        ));
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        
        float milliseconds;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        cublas_total_time += milliseconds;
    }
    
    float cublas_avg_time = cublas_total_time / num_runs;
    double cublas_gflops = calculate_gflops(M, N, K, cublas_avg_time);

    // Copy results back for verification
    CUDA_CHECK(hipMemcpy(h_C_custom.data(), d_C_custom, M * N * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_C_cublas.data(), d_C_cublas, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Print results
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "\nPerformance Results:" << std::endl;
    std::cout << "  Custom Kernel: " << std::setw(8) << custom_avg_time << " ms, " 
              << std::setw(8) << custom_gflops << " GFLOPS" << std::endl;
    std::cout << "  cuBLAS:        " << std::setw(8) << cublas_avg_time << " ms, " 
              << std::setw(8) << cublas_gflops << " GFLOPS" << std::endl;
    std::cout << "  Performance:   " << std::setw(8) << (custom_gflops / cublas_gflops * 100.0) << "% of cuBLAS performance" << std::endl;

    // Verify results
    std::cout << "\nVerification (Custom vs cuBLAS): ";
    if (verify_results(h_C_custom.data(), h_C_cublas.data(), M * N, 1e-3)) {
        std::cout << "PASS" << std::endl;
    } else {
        std::cout << "FAIL" << std::endl;
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C_custom));
    CUDA_CHECK(hipFree(d_C_cublas));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
}

int main() {
    // Print device information
    int device;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    
    std::cout << "GPU: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Shared Memory per Block: " << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
    std::cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Memory Clock Rate: " << prop.memoryClockRate / 1000 << " MHz" << std::endl;
    std::cout << "Memory Bus Width: " << prop.memoryBusWidth << " bits" << std::endl;

    // Test different matrix sizes
    std::vector<unsigned int> sizes = {512, 1024, 2048, 4096, 8192};
    
    for (unsigned int size : sizes) {
        try {
            benchmark_gemm(size, size, size);
        } catch (const std::exception& e) {
            std::cerr << "Error benchmarking size " << size << ": " << e.what() << std::endl;
        }
    }
    
    std::cout << "\n" << std::string(80, '=') << std::endl;
    std::cout << "Benchmark completed!" << std::endl;
    std::cout << std::string(80, '=') << std::endl;

    return 0;
}